#include "hip/hip_runtime.h"
#include <iostream>
#include <GL/glew.h>
#include <GLFW/glfw3.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <unistd.h>
#include <cuda_gl_interop.h>
#include "graphicsIncludes.h"
#include "vendorIncludes.h"
#include "Renderer.h"
#include "Grid.h"
#include "Camera.h"
#include "VideoPlayer.h"
#include "Input.h"
#include "GUI.h"
#include "Window.h"
#include <stdio.h>
#include <stdlib.h>


//CUDA
const int N = 262144;

// camera
Camera camera(glm::vec3(70.0f, 30.0f, 260.0f));
// timing
float deltaTime = 0.0f;	
float lastFrame = 0.0f;
const int width = 1600;
const int height = 1000;
float lastX = width/2.0f;
float lastY = height/2.0f;
bool firstMouse= true;

void mouse_callback(GLFWwindow* window, double xpos, double ypos){
    if (firstMouse){
        lastX = xpos;
        lastY = ypos;
        firstMouse = false;
    }

    float xoffset = xpos - lastX;
    float yoffset = lastY - ypos; // reversed since y-coordinates go from bottom to top
    lastX = xpos;
    lastY = ypos;

    camera.ProcessMouseMovement(xoffset, yoffset);
}


int main(void){

	Window window(width, height, mouse_callback);
	glm::mat4 proj;
	proj = glm::perspective(glm::radians(45.0f), (float)width/(float)height, 0.1f, 1400.0f); //fov, aspect, near, far
	glm::mat4 view = camera.GetViewMatrix();
	glm::mat4 model = glm::mat4(1.0f);//glm::rotate(glm::mat4(1.0f), glm::radians(-100.0f), glm::vec3(1.0f,0.0f,0.0f));
	glm::mat4 mvp = proj * view * model;
	Renderer renderer;	
	Grid grid(0.01f, mvp);
	//GUI gui(window.ptr);
	//set_initail_conditions();

	unsigned int index[N];
	for(unsigned int i = 0; i < N; i++){
		index[i] = i;
	}
	float particles_CPU[N*4];
	
	VertexArray va;
	VertexBuffer vb(N);
	VertexBufferLayout layout;		
	layout.Push<float>(4);
	va.AddBuffer(vb, layout);
	IndexBuffer ib(index, N);
	Shader shader("../res/shaders/particle.shader");
	

	FILE *posFile = fopen("../res/PosAndVel","rb");
	//VideoPlayer vp;	 //TODO: Add file name to constructor, Move functionality noted below into this clas

	float fileTime;  //TODO: MOVE
	//FILE *posFile;
	bool isRead = true;
    while(!window.shouldClose()){
        float currentFrame = glfwGetTime();
        deltaTime = currentFrame - lastFrame;
		lastFrame = currentFrame;
		//gui.NewFrame();
		isRead = Input::processInput(window.ptr, camera, deltaTime, isRead);
		glm::mat4 view = camera.GetViewMatrix();
		mvp = proj*view*model;
		if (isRead){ //TODO: MOVE
			
			fread(&fileTime, sizeof(float), 1,posFile);
			fread(particles_CPU,sizeof(float4),N,posFile);
		}

		renderer.Clear();
		
		if (isRead) {
		  vb.Update(particles_CPU);
		}
		
		
		ib.Bind();
		shader.Bind();
		shader.SetUniformMat4f("u_MVP", mvp);
		renderer.Draw(va,ib,shader,GL_POINTS);
		
		if (isRead)
			fread(particles_CPU,sizeof(float4),N,posFile);
		
		grid.Update(mvp);	
		renderer.Draw(grid);

		//gui.CameraWindowUpdate(camera);
		//gui.Render();
		window.Update();
    }
    return 0;
}
